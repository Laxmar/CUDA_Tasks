#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "utility.h"


hipError_t bubble_sort(float* array, unsigned int size);


__global__ void bubble_sort_kernel(float* dev_array, unsigned int s, unsigned int size)
{
    // s - even/odd stage indicator
    int i, j;
    float a, b;
    int index = 2 * (threadIdx.x + blockDim.x * blockIdx.x);

    i = index + s; 
    j = i + 1;
    if (j < size) 
    { 
        a = dev_array[i];
        b = dev_array[j];
        if (b < a) 
        { 
            dev_array[i] = b;
            dev_array[j] = a;
        } 
    }
}

int main()
{
    const int array_size = 6;
    float a[array_size] = {};

    generate_array(a, array_size, -10, 10);
    //print_array(a, array_size);

    hipError_t cudaStatus = bubble_sort(a, array_size);

    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "bubble_sort failed!");
        return 1;
    }
    print_array(a, array_size);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    printf("Program finished without errros");

    return 0;
}


hipError_t bubble_sort(float* array, unsigned int array_size)
{
    float* dev_array = 0;
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_array, array_size * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_array, array, array_size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // launch Kernel
    for (unsigned int i = 0; i < array_size - 1; i++) 
    { 
        bubble_sort_kernel<<<1, array_size>>> (dev_array, (i % 2), array_size);
       // kernel << <blocks, threads >> >
    }

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(array, dev_array, array_size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_array);

    return cudaStatus;
}